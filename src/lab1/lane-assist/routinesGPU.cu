#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <hip/hip_runtime.h>
#include <algorithm>
#include "png_io.h"
#include "routinesGPU.h"

#define BLOCK_SIZE 32

__global__ void noiseReduction(uint8_t* im, float* NR, int width, int height) {
	int i = threadIdx.y + blockIdx.y * blockDim.y;
	int j = threadIdx.x + blockIdx.x * blockDim.x;

	if((i < 2 || i > height - 3) || (j < 2 || j > width - 3)) return;

	// Noise reduction
	NR[i*width+j] =
	(2.0*im[(i-2)*width+(j-2)] +  4.0*im[(i-2)*width+(j-1)] +  5.0*im[(i-2)*width+(j)] +  4.0*im[(i-2)*width+(j+1)] + 2.0*im[(i-2)*width+(j+2)]
	+ 4.0*im[(i-1)*width+(j-2)] +  9.0*im[(i-1)*width+(j-1)] + 12.0*im[(i-1)*width+(j)] +  9.0*im[(i-1)*width+(j+1)] + 4.0*im[(i-1)*width+(j+2)]
	+ 5.0*im[(i  )*width+(j-2)] + 12.0*im[(i  )*width+(j-1)] + 15.0*im[(i  )*width+(j)] + 12.0*im[(i  )*width+(j+1)] + 5.0*im[(i  )*width+(j+2)]
	+ 4.0*im[(i+1)*width+(j-2)] +  9.0*im[(i+1)*width+(j-1)] + 12.0*im[(i+1)*width+(j)] +  9.0*im[(i+1)*width+(j+1)] + 4.0*im[(i+1)*width+(j+2)]
	+ 2.0*im[(i+2)*width+(j-2)] +  4.0*im[(i+2)*width+(j-1)] +  5.0*im[(i+2)*width+(j)] +  4.0*im[(i+2)*width+(j+1)] + 2.0*im[(i+2)*width+(j+2)])
	/159.0;
}

__global__ void gradient(float* G, float* Gx, float* Gy, float* NR, float* phi, int width, int height) {
	int i = threadIdx.y + blockIdx.y * blockDim.y;
	int j = threadIdx.x + blockIdx.x * blockDim.x;

	if((i < 2 || i > height - 3) || (j < 2 || j > width - 3)) return;

	float PI = 3.141593;
	
	// Intensity gradient of the image
	Gx[i*width+j] = 
		(1.0*NR[(i-2)*width+(j-2)] +  2.0*NR[(i-2)*width+(j-1)] +  (-2.0)*NR[(i-2)*width+(j+1)] + (-1.0)*NR[(i-2)*width+(j+2)]
		+ 4.0*NR[(i-1)*width+(j-2)] +  8.0*NR[(i-1)*width+(j-1)] +  (-8.0)*NR[(i-1)*width+(j+1)] + (-4.0)*NR[(i-1)*width+(j+2)]
		+ 6.0*NR[(i  )*width+(j-2)] + 12.0*NR[(i  )*width+(j-1)] + (-12.0)*NR[(i  )*width+(j+1)] + (-6.0)*NR[(i  )*width+(j+2)]
		+ 4.0*NR[(i+1)*width+(j-2)] +  8.0*NR[(i+1)*width+(j-1)] +  (-8.0)*NR[(i+1)*width+(j+1)] + (-4.0)*NR[(i+1)*width+(j+2)]
		+ 1.0*NR[(i+2)*width+(j-2)] +  2.0*NR[(i+2)*width+(j-1)] +  (-2.0)*NR[(i+2)*width+(j+1)] + (-1.0)*NR[(i+2)*width+(j+2)]);


	Gy[i*width+j] = 
		((-1.0)*NR[(i-2)*width+(j-2)] + (-4.0)*NR[(i-2)*width+(j-1)] +  (-6.0)*NR[(i-2)*width+(j)] + (-4.0)*NR[(i-2)*width+(j+1)] + (-1.0)*NR[(i-2)*width+(j+2)]
		+ (-2.0)*NR[(i-1)*width+(j-2)] + (-8.0)*NR[(i-1)*width+(j-1)] + (-12.0)*NR[(i-1)*width+(j)] + (-8.0)*NR[(i-1)*width+(j+1)] + (-2.0)*NR[(i-1)*width+(j+2)]
		+    2.0*NR[(i+1)*width+(j-2)] +    8.0*NR[(i+1)*width+(j-1)] +    12.0*NR[(i+1)*width+(j)] +    8.0*NR[(i+1)*width+(j+1)] +    2.0*NR[(i+1)*width+(j+2)]
		+    1.0*NR[(i+2)*width+(j-2)] +    4.0*NR[(i+2)*width+(j-1)] +     6.0*NR[(i+2)*width+(j)] +    4.0*NR[(i+2)*width+(j+1)] +    1.0*NR[(i+2)*width+(j+2)]);

	G[i*width+j]   = sqrtf((Gx[i*width+j]*Gx[i*width+j])+(Gy[i*width+j]*Gy[i*width+j]));	//G = √Gx²+Gy²
	phi[i*width+j] = atan2f(fabs(Gy[i*width+j]),fabs(Gx[i*width+j]));

	if(fabs(phi[i*width+j])<=PI/8 )
		phi[i*width+j] = 0;
	else if (fabs(phi[i*width+j])<= 3*(PI/8))
		phi[i*width+j] = 45;
	else if (fabs(phi[i*width+j]) <= 5*(PI/8))
		phi[i*width+j] = 90;
	else if (fabs(phi[i*width+j]) <= 7*(PI/8))
		phi[i*width+j] = 135;
	else phi[i*width+j] = 0;
	
}

__global__ void edgeDetection(uint8_t* pedge, float* G, float* phi, int width, int height) {
	int i = threadIdx.y + blockIdx.y * blockDim.y;
	int j = threadIdx.x + blockIdx.x * blockDim.x;

	if((i < 3 || i > height - 4) || (j < 3 || j > width - 4)) return;

	if(phi[i*width+j] == 0) {
		if(G[i*width+j]>G[i*width+j+1] && G[i*width+j]>G[i*width+j-1]) //edge is in N-S
			pedge[i*width+j] = 1;
	} else if(phi[i*width+j] == 45) {
		if(G[i*width+j]>G[(i+1)*width+j+1] && G[i*width+j]>G[(i-1)*width+j-1]) // edge is in NW-SE
			pedge[i*width+j] = 1;

	} else if(phi[i*width+j] == 90) {
		if(G[i*width+j]>G[(i+1)*width+j] && G[i*width+j]>G[(i-1)*width+j]) //edge is in E-W
			pedge[i*width+j] = 1;

	} else if(phi[i*width+j] == 135) {
		if(G[i*width+j]>G[(i+1)*width+j-1] && G[i*width+j]>G[(i-1)*width+j+1]) // edge is in NE-SW
			pedge[i*width+j] = 1;
	}
	
}

__global__ void hysteresis(uint8_t* image_out, uint8_t* pedge, float* G, int width, int height, float level) {
	int i = threadIdx.y + blockIdx.y * blockDim.y;
	int j = threadIdx.x + blockIdx.x * blockDim.x;
	
	if((i < 3 || i > height - 4) || (j < 3 || j > width - 4)) return;
	
	int ii, jj;
	float lowthres = level/2, hithres = 2 *(level);

	if(G[i*width+j]>hithres && pedge[i*width+j]) {
		image_out[i*width+j] = 255;
	}
	else if(pedge[i*width+j] && G[i*width+j]>=lowthres && G[i*width+j]<hithres) {
		// check neighbours 3x3
		for (ii=-1;ii<=1; ii++)
			for (jj=-1;jj<=1; jj++)
				if (G[(i+ii)*width+j+jj]>hithres) {
					image_out[i*width+j] = 255;
				}
	}
	
}

__global__ void houghKernel(uint8_t* im, uint32_t* accumulators, int width, int height, int accu_width, int accu_height, 
	float* sin_table, float* cos_table) 
{
	int theta;

	int i = threadIdx.y + blockIdx.y * blockDim.y;
	int j = threadIdx.x + blockIdx.x * blockDim.x;

	if(i >= height || j >= width) return;

	float hough_h = ((sqrt(2.0) * (float)(height>width?height:width)) / 2.0);

	float center_x = width/2.0; 
	float center_y = height/2.0;

	if( im[ (i*width) + j] > 250 ) // Pixel is edge  
	{  
		for(theta=0;theta<180;theta++)  
		{  
			float rho = ( ((float)j - center_x) * cos_table[theta]) + (((float)i - center_y) * sin_table[theta]);
			atomicAdd(&accumulators[(int)((round(rho + hough_h) * 180.0)) + theta], 1);
		} 
	} 
	
}

__global__ void getLinesKernel(int threshold, uint32_t* accumulators, int accu_width, int accu_height, int width, int height, 
	float* sin_table, float* cos_table, int* x1_lines, int* y1_lines, int* x2_lines, int* y2_lines, int* lines) 
{
	int rho = threadIdx.y + blockIdx.y * blockDim.y;
	int theta = threadIdx.x + blockIdx.x * blockDim.x;

	if(rho >= accu_height || theta >= accu_width) return;

	uint32_t max;

	if(accumulators[(rho*accu_width) + theta] >= threshold)  
	{  
		//Is this point a local maxima (9x9)  
		max = accumulators[(rho*accu_width) + theta]; 
		for(int ii=-4;ii<=4;ii++)  
		{  
			for(int jj=-4;jj<=4;jj++)  
			{  
				if( (ii+rho>=0 && ii+rho<accu_height) && (jj+theta>=0 && jj+theta<accu_width) )  
				{  
					if( accumulators[((rho+ii)*accu_width) + (theta+jj)] > max )  
					{
						max = accumulators[((rho+ii)*accu_width) + (theta+jj)];
					}  
				}  
			}  
		}  

		if(max == accumulators[(rho*accu_width) + theta]) //local maxima
		{
			int x1, y1, x2, y2;  
			x1 = y1 = x2 = y2 = 0;  

			if(theta >= 45 && theta <= 135)  
			{
				if (theta>90) {
					//y = (r - x cos(t)) / sin(t)  
					x1 = width/2;  
					y1 = ((float)(rho-(accu_height/2)) - ((x1 - (width/2) ) * cos_table[theta])) / sin_table[theta] + (height / 2);
					x2 = width;  
					y2 = ((float)(rho-(accu_height/2)) - ((x2 - (width/2) ) * cos_table[theta])) / sin_table[theta] + (height / 2);  
				} else {
					//y = (r - x cos(t)) / sin(t)  
					x1 = 0;  
					y1 = ((float)(rho-(accu_height/2)) - ((x1 - (width/2) ) * cos_table[theta])) / sin_table[theta] + (height / 2);
					x2 = width*2/5;  
					y2 = ((float)(rho-(accu_height/2)) - ((x2 - (width/2) ) * cos_table[theta])) / sin_table[theta] + (height / 2); 
				}
			} else {
				//x = (r - y sin(t)) / cos(t);  
				y1 = 0;  
				x1 = ((float)(rho-(accu_height/2)) - ((y1 - (height/2) ) * sin_table[theta])) / cos_table[theta] + (width / 2);  
				y2 = height;  
				x2 = ((float)(rho-(accu_height/2)) - ((y2 - (height/2) ) * sin_table[theta])) / cos_table[theta] + (width / 2);  
			}
			x1_lines[*lines] = x1;
			y1_lines[*lines] = y1;
			x2_lines[*lines] = x2;
			y2_lines[*lines] = y2;
			atomicAdd(lines, 1);
		}
	}
		
	
}

void canny(uint8_t *im, uint8_t *image_out, int height, int width, float level) 
{
	float* NR, *G, *Gx, *Gy, *phi;
	uint8_t* imTmp, *pedge, *imageoutTmp;

	int xSum = 0, ySum = 0;
	dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
	if(width % dimBlock.x != 0) xSum = 1;
	if(height % dimBlock.y != 0) ySum = 1;
	dim3 dimGrid((width / dimBlock.x) + xSum, (height / dimBlock.y) + ySum);

	// Pasamos a memoria la imagen y un NR temporal
	hipMalloc((void**)&NR, width * height * sizeof(float));
	hipMalloc((void**)&imTmp, width * height * sizeof(uint8_t));
	hipMemcpy(imTmp, im, width * height * sizeof(uint8_t), hipMemcpyHostToDevice);
	noiseReduction<<<dimGrid,dimBlock>>>(imTmp, NR, width, height);
	hipDeviceSynchronize();
	// En cuanto dejemos de usar la imagen, la liberamos de memoria
	hipFree(imTmp);

	// Reservamos memoria para el resto de variables que vamos a utilizar
	hipMalloc((void**)&G, width * height * sizeof(float));
	hipMalloc((void**)&Gx, width * height * sizeof(float));
	hipMalloc((void**)&Gy, width * height * sizeof(float));
	hipMalloc((void**)&phi, width * height * sizeof(float));
	gradient<<<dimGrid,dimBlock>>>(G, Gx, Gy, NR, phi, width, height);
	hipDeviceSynchronize();
	// Liberamos las que ya no vayamos a utilizar
	hipFree(NR);
	hipFree(Gx);
	hipFree(Gy);

	// Reservamos el pedge
	hipMalloc((void**)&pedge, width * height * sizeof(uint8_t));
	hipMemset(pedge, 0, width * height * sizeof(uint8_t));
	edgeDetection<<<dimGrid,dimBlock>>>(pedge, G, phi, width, height);
	hipDeviceSynchronize();
	// Y liberamos phi que ya no se va a usar
	hipFree(phi);

	// Reservamos memoria para la imagen final
	hipMalloc((void**)&imageoutTmp, width * height * sizeof(uint8_t));
	hipMemset(imageoutTmp, 0, width * height * sizeof(uint8_t));
	hysteresis<<<dimGrid,dimBlock>>>(imageoutTmp, pedge, G, width, height, level);
	hipDeviceSynchronize();
	// Y la pasamos a memoria física una vez la hayamos calculado.
	hipMemcpy(image_out, imageoutTmp, width * height * sizeof(uint8_t), hipMemcpyDeviceToHost);

	hipFree(G);
	hipFree(pedge);
	hipFree(imageoutTmp);
}

void houghTransform(uint8_t* im, uint32_t* accumulators, int width, int height, int accu_width, int accu_height, 
	float* sin_table, float* cos_table) 
{
	uint8_t* imTmp;
	uint32_t* accuTmp;
	float* sinTTmp, *cosTTmp;

	int xSum = 0, ySum = 0;
	dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
	if(width % dimBlock.x != 0) xSum = 1;
	if(height % dimBlock.y != 0) ySum = 1;
	dim3 dimGrid((width / dimBlock.x) + xSum, (height / dimBlock.y) + ySum);

	hipMalloc((void**)&imTmp, width * height * sizeof(uint8_t));
	hipMalloc((void**)&accuTmp, accu_width * accu_height * sizeof(uint32_t));
	hipMalloc((void**)&sinTTmp, 180 * sizeof(float));
	hipMalloc((void**)&cosTTmp, 180 * sizeof(float));
	
	hipMemcpy(imTmp, im, width * height * sizeof(uint8_t), hipMemcpyHostToDevice);
	hipMemcpy(sinTTmp, sin_table, 180 * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(cosTTmp, cos_table, 180 * sizeof(float), hipMemcpyHostToDevice);
	hipMemset(accuTmp, 0, accu_width * accu_height * sizeof(uint32_t));

	houghKernel<<<dimGrid, dimBlock>>>(imTmp, accuTmp, width, height, accu_width, accu_height, sinTTmp, cosTTmp);
	hipDeviceSynchronize();
	hipMemcpy(accumulators, accuTmp, accu_width * accu_height * sizeof(uint32_t), hipMemcpyDeviceToHost);

	hipFree(sinTTmp);
	hipFree(cosTTmp);
	hipFree(imTmp);
	hipFree(accuTmp);
}

void getLines(int threshold, uint32_t* accumulators, int accu_width, int accu_height, int width, int height, 
	float* sin_table, float* cos_table, int* x1, int* y1, int* x2, int* y2, int* lines) 
{
	uint32_t* accuTmp;
	float* sinTTmp, *cosTTmp;
	int* x1Tmp, *y1Tmp, *x2Tmp, *y2Tmp, *linesTmp;

	int xSum = 0, ySum = 0;
	dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
	int maximumWidth = std::max(width, accu_width);
	int maximumHeight = std::max(height, accu_height);
	if(maximumWidth % dimBlock.x != 0) xSum = 1;
	if(maximumHeight % dimBlock.y != 0) ySum = 1;
	dim3 dimGrid((maximumWidth / dimBlock.x) + xSum, (maximumHeight / dimBlock.y) + ySum);

	hipMalloc((void**)&accuTmp, accu_width * accu_height * sizeof(uint32_t));
	hipMalloc((void**)&sinTTmp, 180 * sizeof(float));
	hipMalloc((void**)&cosTTmp, 180 * sizeof(float));
	hipMalloc((void**)&x1Tmp, 10 * sizeof(int));
	hipMalloc((void**)&y1Tmp, 10 * sizeof(int));
	hipMalloc((void**)&x2Tmp, 10 * sizeof(int));
	hipMalloc((void**)&y2Tmp, 10 * sizeof(int));
	hipMalloc((void**)&linesTmp, sizeof(int));

	hipMemcpy(accuTmp, accumulators, accu_width * accu_height * sizeof(uint32_t), hipMemcpyHostToDevice);
	hipMemcpy(sinTTmp, sin_table, 180 * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(cosTTmp, cos_table, 180 * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(linesTmp, lines, sizeof(int), hipMemcpyHostToDevice);
	
	getLinesKernel<<<dimGrid, dimBlock>>>(threshold, accuTmp, accu_width, accu_height, width, height, sinTTmp, cosTTmp, x1Tmp, y1Tmp, x2Tmp, y2Tmp, linesTmp);
	hipDeviceSynchronize();

	hipMemcpy(x1, x1Tmp, 10 * sizeof(int), hipMemcpyDeviceToHost);
	hipMemcpy(y1, y1Tmp, 10 * sizeof(int), hipMemcpyDeviceToHost);
	hipMemcpy(x2, x2Tmp, 10 * sizeof(int), hipMemcpyDeviceToHost);
	hipMemcpy(y2, y2Tmp, 10 * sizeof(int), hipMemcpyDeviceToHost);
	hipMemcpy(lines, linesTmp, sizeof(int), hipMemcpyDeviceToHost);

	hipFree(accuTmp);
	hipFree(sinTTmp);
	hipFree(cosTTmp);
	hipFree(x1Tmp);
	hipFree(y1Tmp);
	hipFree(x2Tmp);
	hipFree(y2Tmp);
	hipFree(linesTmp);
}

void lane_assist_GPU(uint8_t *im, int height, int width,
	uint8_t *imEdge, float *sin_table, float *cos_table, 
	uint32_t *accum, int accu_height, int accu_width,
	int *x1, int *y1, int *x2, int *y2, int *nlines)
{
	float level = 1000.0f;
	int threshold = width > height ? width/6 : height / 6;

	canny(im, imEdge, height, width, level);
	write_png_fileBW("out_edges.png", imEdge, width, height);

	houghTransform(imEdge, accum, width, height, accu_width, accu_height, sin_table, cos_table);

	getLines(threshold, accum, accu_width, accu_height, width, height, sin_table, cos_table, x1, y1, x2, y2, nlines);
}
